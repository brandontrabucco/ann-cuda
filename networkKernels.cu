#include "hip/hip_runtime.h"
#include "networkKernels.cuh"

vector<int> factor(int f) {
	vector<int> factors = {1, 1};
	bool s = false;
    for(int ii = 2; ii<=f; ii++) {
        while(f % ii == 0) {
            f = f/ii;
            if (s) {
            	factors[0] *= ii ;
            	s = !s;
            } else {
            	factors[1] *= ii ;
            	s = !s;
            }
        }
    } return factors;
}

__global__ void inputNeuronKernel(double *input, PassiveNeuron nodes[], double scalar, double *output, int nNeurons, int nPerThread, int nThreads) {
	for (int i = 0; i < nPerThread; i++) {
		int neuronId = threadIdx.x + (i * nThreads);
		if (neuronId < nNeurons) output[neuronId] = nodes[neuronId].get(input[neuronId], scalar);
	}
}

__global__ void neuronKernel(double *input, Neuron nodes[], double *output, int nNeurons, int nPerThread, int nThreads) {
	for (int i = 0; i < nPerThread; i++) {
		int neuronId = threadIdx.x + (i * nThreads);
		if (neuronId < nNeurons) output[neuronId] = nodes[neuronId].get(input[neuronId]);
	}
}

__global__ void synapseKernel(double *input, Synapse connections[], double *output, int nNeuronsCurrent, int nNeuronsPrevious, int nPerThread, int nPerBlock, int nThreads, int nBlocks) {
	for (int i = 0; i < nPerBlock; i++) {
		for (int j = 0; j < nPerThread; j++) {
			int blockId = blockIdx.x + (i * nBlocks);	// the current layer
			int threadId = threadIdx.x + (j * nThreads);	// the previous layer
			if (blockId < nNeuronsCurrent && threadId < nNeuronsPrevious) output[blockId * (nNeuronsPrevious) + threadId] = connections[blockId * (nNeuronsPrevious) + threadId].get(input[threadId]);
			//printf("Input = %f : Output = %f\n", input[threadId], output[blockId * (blockDim.x * blockDim.y) + threadId]);
		}
	}
}

__global__ void sumInputKernel(double *input, double *output, int nConnectionsPer, int nNeurons, int nPerThread, int nThreads) {
	for (int i = 0; i < nPerThread; i++) {
		int neuronId = threadIdx.x + (i * nThreads);
		if (neuronId < nNeurons) for (int j = 0; j < nConnectionsPer; j++) {
			output[neuronId] +=	input[(neuronId * nConnectionsPer) + j];
			//if (neuronId == 0)printf("From %f\n", input[(neuronId * nConnections) + j]);
		}
	}
}

__global__ void hiddenLayerGradientDescentKernel(double *errorPrime, double learningRate, Neuron nodes[], Neuron previous[], Synapse connections[], int nNeuronsCurrent, int nNeuronsPrevious, int nPerThread, int nPerBlock, int nThreads, int nBlocks) {
	for (int i = 0; i < nPerBlock; i++) {
		for (int j = 0; j < nPerThread; j++) {
			int blockId = blockIdx.x + (i * nBlocks);	// the current layer
			int threadId = threadIdx.x + (j * nThreads);	// the previous layer
			if (blockId < nNeuronsCurrent && threadId < nNeuronsPrevious) 	connections[blockId * (nNeuronsPrevious) + threadId].weight -= learningRate * errorPrime[blockId] * nodes[blockId].derivative * previous[threadId].activation;
		}
	}
}

__global__ void outputLayerGradientDescentKernel(double *errorPrime, double learningRate, Neuron previous[], Synapse connections[], int nNeuronsCurrent, int nNeuronsPrevious, int nPerThread, int nPerBlock, int nThreads, int nBlocks) {
	for (int i = 0; i < nPerBlock; i++) {
		for (int j = 0; j < nPerThread; j++) {
			int blockId = blockIdx.x + (i * nBlocks);	// the current layer
			int threadId = threadIdx.x + (j * nThreads);	// the previous layer
			if (blockId < nNeuronsCurrent && threadId < nNeuronsPrevious) 	connections[blockId * (nNeuronsPrevious) + threadId].weight -= learningRate * errorPrime[blockId] * previous[threadId].activation;
		}
	}
}

__global__ void hiddenLayerSumErrorKernel(double *errorPrime, Neuron nodes[], Synapse connections[], double *output, int nNeuronsCurrent, int nNeuronsPrevious, int nPerThread, int nThreads) {
	for (int i = 0; i < nPerThread; i++) {
		int neuronId = threadIdx.x + (i * nThreads);
		if (neuronId < nNeuronsPrevious) for (int j = 0; j < (nNeuronsCurrent); j++) {
			output[neuronId] += ((errorPrime[j] * nodes[neuronId].derivative) * connections[(j * nNeuronsPrevious) + neuronId].weight);
		}
	}
}

__global__ void outputLayerSumErrorKernel(double *errorPrime, Synapse connections[], double *output, int nNeuronsCurrent, int nNeuronsPrevious, int nPerThread, int nThreads) {
	for (int i = 0; i < nPerThread; i++) {
		int neuronId = threadIdx.x + (i * nThreads);
		if (neuronId < nNeuronsPrevious) for (int j = 0; j < (nNeuronsCurrent); j++) {
			output[neuronId] += (errorPrime[j] * connections[(j * nNeuronsPrevious) + neuronId].weight);
		}
	}
}

int KernelAdapter::nPerThread = 0;
int KernelAdapter::nPerBlock = 0;

void KernelAdapter::startInputNeuronKernel(double *input, PassiveNeuron nodes[], double scalar, double *output, int nNeurons) {
	KernelAdapter::nPerThread = (int)(nNeurons / KernelAdapter::nThreads);
	inputNeuronKernel<<<1, KernelAdapter::nThreads>>>(input, nodes, scalar, output, nNeurons, nPerThread, nThreads);
	hipDeviceSynchronize();
}

void KernelAdapter::startNeuronKernel(double *input, Neuron nodes[], double *output, int nNeurons) {
	KernelAdapter::nPerThread = (int)(nNeurons / KernelAdapter::nThreads);
	neuronKernel<<<1, KernelAdapter::nThreads>>>(input, nodes, output, nNeurons, KernelAdapter::nPerThread, KernelAdapter::nThreads);
	hipDeviceSynchronize();
}

void KernelAdapter::startSynapseKernel(double *input, Synapse connections[], double *output, int nNeuronsCurrent, int nNeuronsPrevious) {
	KernelAdapter::nPerThread = (int)(nNeuronsPrevious / KernelAdapter::nThreads);
	KernelAdapter::nPerBlock = (int)(nNeuronsCurrent / KernelAdapter::nBlocks);
	synapseKernel<<<KernelAdapter::nBlocks, KernelAdapter::nThreads>>>(input, connections, output, nNeuronsCurrent, nNeuronsPrevious, KernelAdapter::nPerThread, KernelAdapter::nPerBlock, KernelAdapter::nThreads, KernelAdapter::nBlocks);
	hipDeviceSynchronize();
}

void KernelAdapter::startSumInputKernel(double *input, double *output, int nConnectionsPer, int nNeurons) {
	KernelAdapter::nPerThread = (int)(nNeurons / KernelAdapter::nThreads);
	sumInputKernel<<<1, KernelAdapter::nThreads>>>(input, output, nConnectionsPer, nNeurons, KernelAdapter::nPerThread, KernelAdapter::nThreads);
	hipDeviceSynchronize();
}

void KernelAdapter::startHiddenLayerGradientDescentKernel(double *errorPrime, double learningRate, Neuron nodes[], Neuron previous[], Synapse connections[], int nNeuronsCurrent, int nNeuronsPrevious) {
	KernelAdapter::nPerThread = (int)(nNeuronsPrevious / KernelAdapter::nThreads);
	KernelAdapter::nPerBlock = (int)(nNeuronsCurrent / KernelAdapter::nBlocks);
	hiddenLayerGradientDescentKernel<<<KernelAdapter::nBlocks, KernelAdapter::nThreads>>>(errorPrime, learningRate, nodes, previous, connections, nNeuronsCurrent, nNeuronsPrevious, KernelAdapter::nPerThread, KernelAdapter::nPerBlock, KernelAdapter::nThreads, KernelAdapter::nBlocks);
	hipDeviceSynchronize();
}

void KernelAdapter::startOutputLayerGradientDescentKernel(double *errorPrime, double learningRate, Neuron previous[], Synapse connections[], int nNeuronsCurrent, int nNeuronsPrevious) {
	KernelAdapter::nPerThread = (int)(nNeuronsPrevious / KernelAdapter::nThreads);
	KernelAdapter::nPerBlock = (int)(nNeuronsCurrent / KernelAdapter::nBlocks);
	outputLayerGradientDescentKernel<<<KernelAdapter::nBlocks, KernelAdapter::nThreads>>>(errorPrime, learningRate, previous, connections, nNeuronsCurrent, nNeuronsPrevious, KernelAdapter::nPerThread, KernelAdapter::nPerBlock, KernelAdapter::nThreads, KernelAdapter::nBlocks);
	hipDeviceSynchronize();
}

void KernelAdapter::startHiddenLayerSumErrorKernel(double *errorPrime, Neuron nodes[], Synapse connections[], double *output, int nConnectionsPer, int nNeurons) {
	KernelAdapter::nPerThread = (int)(nNeurons / KernelAdapter::nThreads);
	hiddenLayerSumErrorKernel<<<1, KernelAdapter::nThreads>>>(errorPrime, nodes, connections, output, nConnectionsPer, nNeurons, KernelAdapter::nPerThread, KernelAdapter::nThreads);
	hipDeviceSynchronize();
}

void KernelAdapter::startOutputLayerSumErrorKernel(double *errorPrime, Synapse connections[], double *output, int nNeuronsCurrent, int nNeuronsPrevious) {
	KernelAdapter::nPerThread = (int)(nNeuronsPrevious / KernelAdapter::nThreads);
	outputLayerSumErrorKernel<<<1, KernelAdapter::nThreads>>>(errorPrime, connections, output, nNeuronsCurrent, nNeuronsPrevious, KernelAdapter::nPerThread, KernelAdapter::nThreads);
	hipDeviceSynchronize();
}
