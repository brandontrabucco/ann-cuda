/*
 * HiddenLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "HiddenLayer.cuh"

HiddenLayer::HiddenLayer(int w, int d, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	previousLayerNeurons = d;

	// add neurons and synapses to this layer
	for (int i = 0; i < w; i++) {
		Neuron n = Neuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Neuron created " << i << endl;
	} for (int i = 0; i < (w * d); i++) {
		Synapse s = Synapse();
		s.index = i;
		synapses.push_back(s);
		if (debug) cout << "Synapse created " << i << endl;
	}
}

HiddenLayer::~HiddenLayer() {
	// TODO Auto-generated destructor stub
}

// parallelize each synapse and neuron

vector<double> HiddenLayer::feedforward(vector<double> input) {
	vector<double> output(currentLayerNeurons);	// variables to store data for math operations

	double *deviceInput, *deviceOutput, *deviceSum, *deviceActivation;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons;

	// copy memory to device
	int status;
	long long startTime = getNanoSec();
	if ((status = hipMalloc((void **)&deviceInput, (input.size() * sizeof(double)))) != 0) cout << "error h-1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceOutput, (synapses.size() * sizeof(double)))) != 0) cout << "error h-2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSum, (neurons.size() * sizeof(double)))) != 0) cout << "error h-3 " << status << endl;
	if ((status = hipMalloc((void **)&deviceActivation, (neurons.size() * sizeof(double)))) != 0) cout << "error h-4 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error h-5 " << status << endl;
	if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)))) != 0) cout << "error h-6 " << status << endl;

	if ((status = hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error h-7 " << status << endl;
	if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error h-8 " << status << endl;
	if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error h-9 " << status << endl;
	if ((status = hipMemset(&deviceSum[0], 0, (neurons.size() * sizeof(double)))) != 0) cout << "error h-10 " << status << endl;
	NeuralNetwork::overhead += (getNanoSec() - (startTime));

	startTime = getNanoSec();
	KernelAdapter::startSynapseKernel(deviceInput, deviceSynapses, deviceOutput, currentLayerNeurons, previousLayerNeurons);
	KernelAdapter::startSumInputKernel(deviceOutput, deviceSum, currentLayerNeurons, previousLayerNeurons);
	KernelAdapter::startNeuronKernel(deviceSum, deviceNeurons, deviceActivation, currentLayerNeurons);
	NeuralNetwork::computation += (getNanoSec() - (startTime));

	// get the output from the device
	startTime = getNanoSec();
	if ((status = hipMemcpy(&output[0], &deviceActivation[0],(neurons.size() * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error h-11 " << status << endl;
	if ((status = hipMemcpy(&neurons[0], &deviceNeurons[0],(neurons.size() * sizeof(Neuron)), hipMemcpyDeviceToHost)) != 0) cout << "error h-12 " << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceInput)) != 0) cout << "error h-13 " << status << endl;
	if ((status = hipFree(deviceOutput)) != 0) cout << "error h-14 " << status << endl;
	if ((status = hipFree(deviceSum)) != 0) cout << "error h-15 " << status << endl;
	if ((status = hipFree(deviceActivation)) != 0) cout << "error h-16 " << status << endl;
	if ((status = hipFree(deviceSynapses)) != 0) cout << "error h-17 " << status << endl;
	if ((status = hipFree(deviceNeurons)) != 0) cout << "error h-18 " << status << endl;
	hipDeviceSynchronize();
	NeuralNetwork::overhead += (getNanoSec() - (startTime));

	return output;
}

// parallelize each synapse update

vector<double> HiddenLayer::backpropagate(vector<double> error, double learningRate, vector<Neuron> previousLayer) {
	// iterate through each synapse connected to the previous layer
		vector<double> sum(previousLayerNeurons);	// must be initialized to be read and write

		double *deviceError, *deviceSum;
		Synapse *deviceSynapses;
		Neuron *deviceNeurons, *devicePreviousLayer;

		int status;
		long long startTime = getNanoSec();
		if ((status = hipMalloc((void **)&deviceError, (error.size() * sizeof(double)))) != 0) cout << "error h-1 " << status << endl;
		if ((status = hipMalloc((void **)&deviceSum, (previousLayerNeurons * sizeof(double)))) != 0) cout << "error h-2 " << status << endl;
		if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error h-4 " << status << endl;
		if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)))) != 0) cout << "error h-5 " << status << endl;
		if ((status = hipMalloc((void **)&devicePreviousLayer, (previousLayer.size() * sizeof(Neuron)))) != 0) cout << "error h-6 " << status << endl;

		if ((status = hipMemcpy(&deviceError[0], &error[0], (error.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error h-7 " << status << endl;
		if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error h-9 " << status << endl;
		if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error h-10 " << status << endl;
		if ((status = hipMemcpy(&devicePreviousLayer[0], &previousLayer[0], (previousLayer.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error h-11 " << status << endl;
		if ((status = hipMemset(&deviceSum[0], 0, (previousLayerNeurons * sizeof(double))) != 0)) cout << "error h-12 " << status << endl;
		NeuralNetwork::overhead += (getNanoSec() - startTime);

		startTime = getNanoSec();
		KernelAdapter::startHiddenLayerGradientDescentKernel(deviceError, learningRate, deviceNeurons, devicePreviousLayer, deviceSynapses, currentLayerNeurons, previousLayerNeurons);
		KernelAdapter::startHiddenLayerSumErrorKernel(deviceError, deviceNeurons, deviceSynapses, deviceSum, currentLayerNeurons, previousLayerNeurons);
		NeuralNetwork::computation += (getNanoSec() - startTime);

		startTime = getNanoSec();
		if ((status = hipMemcpy(&synapses[0], &deviceSynapses[0],(synapses.size() * sizeof(Synapse)), hipMemcpyDeviceToHost)) != 0) cout << "error h-13 " << status << endl;
		if ((status = hipMemcpy(&sum[0], &deviceSum[0],(previousLayerNeurons * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error h-14 " << status << endl;
		hipDeviceSynchronize();

		// release memory from GPU
		if ((status = hipFree(deviceError)) != 0) cout << "error h-15 " << status << endl;
		if ((status = hipFree(deviceSum)) != 0) cout << "error h-16 " << status << endl;
		if ((status = hipFree(deviceSynapses)) != 0) cout << "error h-17 " << status << endl;
		if ((status = hipFree(deviceNeurons)) != 0) cout << "error h-18 " << status << endl;
		if ((status = hipFree(devicePreviousLayer)) != 0) cout << "error h-19 " << status << endl;
		hipDeviceSynchronize();
		NeuralNetwork::overhead += (getNanoSec() - startTime);

		return sum;
}
