/*
 * HiddenLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "HiddenLayer.cuh"

HiddenLayer::HiddenLayer(int w, int d, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	previousLayerNeurons = d;

	vector<int> temp = factor(currentLayerNeurons);
	kernelGridHeight = temp[0];
	kernelGridWidth = temp[1];

	temp = factor(previousLayerNeurons);
	kernelBlockHeight = temp[0];
	kernelBlockWidth = temp[1];

	// add neurons and synapses to this layer
	for (int i = 0; i < w; i++) {
		Neuron n = Neuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Neuron created " << i << endl;
	} for (int i = 0; i < (w * d); i++) {
		Synapse s = Synapse();
		s.index = i;
		synapses.push_back(s);
		if (debug) cout << "Synapse created " << i << endl;
	}
}

HiddenLayer::~HiddenLayer() {
	// TODO Auto-generated destructor stub
}

// parallelize each synapse and neuron

vector<double> HiddenLayer::feedforward(vector<double> input) {
	vector<double> output(currentLayerNeurons);	// variables to store data for math operations

	double *deviceInput, *deviceOutput, *deviceSum, *deviceActivation;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons;

	// copy memory to device
	if (hipMalloc((void **)&deviceInput, (input.size() * sizeof(double))) != 0) cout << "error 1" << endl;
	if (hipMalloc((void **)&deviceOutput, (synapses.size() * sizeof(double))) != 0) cout << "error 2" << endl;
	if (hipMalloc((void **)&deviceSum, (neurons.size() * sizeof(double))) != 0) cout << "error 3" << endl;
	if (hipMalloc((void **)&deviceActivation, (neurons.size() * sizeof(double))) != 0) cout << "error 4" << endl;
	if (hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse))) != 0) cout << "error 5" << endl;
	if (hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron))) != 0) cout << "error 6" << endl;

	if (hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice) != 0) cout << "error 7" << endl;
	if (hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice) != 0) cout << "error 8" << endl;
	if (hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice) != 0) cout << "error 9" << endl;
	if (hipMemset(&deviceSum[0], 0, (neurons.size() * sizeof(double))) != 0) cout << "error 10" << endl;

	hipDeviceSynchronize();
	activateSynapse<<<dim3(kernelGridWidth, kernelGridHeight), dim3(kernelBlockWidth, kernelBlockHeight)>>>(deviceInput, deviceSynapses, deviceOutput);	// a block represents current layer, thread is previous layer
	hipDeviceSynchronize();
	sumInputFromSynapse<<<dim3(1, 1), dim3(kernelGridWidth, kernelGridHeight)>>>(deviceOutput, deviceSum, previousLayerNeurons);
	hipDeviceSynchronize();
	activateNeuron<<<dim3(1, 1), dim3(kernelGridWidth, kernelGridHeight)>>>(deviceSum, deviceNeurons, deviceActivation);
	hipDeviceSynchronize();


	// get the output from the device
	if (hipMemcpy(&output[0], &deviceActivation[0],(neurons.size() * sizeof(double)), hipMemcpyDeviceToHost) != 0) cout << "error __ 11" << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if (hipFree(deviceInput) != 0) cout << "error 12" << endl;
	if (hipFree(deviceOutput) != 0) cout << "error 13" << endl;
	if (hipFree(deviceSum) != 0) cout << "error 14" << endl;
	if (hipFree(deviceActivation) != 0) cout << "error 15" << endl;
	if (hipFree(deviceSynapses) != 0) cout << "error 16" << endl;
	if (hipFree(deviceNeurons) != 0) cout << "error 17" << endl;
	hipDeviceSynchronize();

	return output;
}

// parallelize each synapse update

vector<double> HiddenLayer::backpropagate(vector<double> error, double learningRate, vector<Neuron> previousLayer) {
	// iterate through each synapse connected to the previous layer
	vector<double> sum(previousLayerNeurons);

	double *deviceError, *deviceSum, deviceLearningRate;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons, *devicePreviousLayer;

	hipMalloc((void **)&deviceError, (error.size() * sizeof(double)));
	hipMalloc((void **)&deviceSum, (previousLayerNeurons * sizeof(double)));
	hipMalloc((void **)&deviceLearningRate, sizeof(double));	// this may be a problem since is only one variable not array
	hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)));
	hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)));
	hipMalloc((void **)&devicePreviousLayer, (previousLayer.size() * sizeof(Neuron)));
	hipDeviceSynchronize();

	hipMemcpy(deviceError, &error[0], (error.size() * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(&deviceLearningRate, &learningRate, (error.size() * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(deviceSynapses, &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice);
	hipMemcpy(deviceNeurons, &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice);
	hipMemcpy(devicePreviousLayer, &previousLayer[0], (previousLayer.size() * sizeof(Neuron)), hipMemcpyHostToDevice);
	hipMemset(deviceSum, 0, (previousLayerNeurons * sizeof(double)));
	hipDeviceSynchronize();

	gradientDescent<<<dim3(kernelGridWidth, kernelGridHeight), dim3(kernelBlockWidth, kernelBlockHeight)>>>(deviceError, deviceLearningRate, deviceNeurons, devicePreviousLayer, deviceSynapses);
	hipDeviceSynchronize();
	sumWeightedError<<<dim3(1, 1), dim3(kernelGridWidth, kernelGridHeight)>>>(deviceError, deviceNeurons, deviceSynapses, deviceSum, previousLayerNeurons);

	hipDeviceSynchronize();
	hipMemcpy(&synapses[0], &deviceSynapses[0],(synapses.size() * sizeof(Synapse)), hipMemcpyDeviceToHost);
	hipMemcpy(&sum[0], &deviceSum[0],(previousLayerNeurons * sizeof(double)), hipMemcpyDeviceToHost);
	return sum;
}

