/*
 * InputLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "InputLayer.cuh"

InputLayer::InputLayer(int w, double range, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	scalar = 1 / range;

	for (int i = 0; i < currentLayerNeurons; i++) {
		PassiveNeuron n = PassiveNeuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Passive Neuron " << i << endl;
	}
}

InputLayer::~InputLayer() {
	// TODO Auto-generated destructor stub
}

vector<double> InputLayer::feedforward(vector<double> input) {
	vector<double> output(neurons.size());

	double *deviceInput, *deviceOutput;
	PassiveNeuron *deviceNeurons;

	// copy memory to device
	int status;
	if ((status = hipMalloc((void **)&deviceInput, (input.size() * sizeof(double)))) != 0) cout << "error h-1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceOutput, (neurons.size() * sizeof(double)))) != 0) cout << "error h-2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(PassiveNeuron)))) != 0) cout << "error h-4 " << status << endl;

	if ((status = hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error h-5 " << status << endl;
	if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(PassiveNeuron)), hipMemcpyHostToDevice)) != 0) cout << "error h-7 " << status << endl;

	// start cuda kernel
	KernelAdapter::startInputNeuronKernel(deviceInput, deviceNeurons, scalar, deviceOutput, currentLayerNeurons);

	// get the output from the device
	if ((status = hipMemcpy(&output[0], &deviceOutput[0], (neurons.size() * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error__ " << status << endl;
	if ((status = hipMemcpy(&neurons[0], &deviceNeurons[0],(neurons.size() * sizeof(PassiveNeuron)), hipMemcpyDeviceToHost)) != 0) cout << "error _*_ h-" << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceInput)) != 0) cout << "error h-8 " << status << endl;
	if ((status = hipFree(deviceOutput)) != 0) cout << "error h-9 " << status << endl;
	if ((status = hipFree(deviceNeurons)) != 0) cout << "error** h-10 " << status << endl;
	hipDeviceSynchronize();

	return output;
}

vector<double> InputLayer::backpropagate(vector<double> error, double learningRate) {
	return error;
}

