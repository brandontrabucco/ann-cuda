/*
 * InputLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "InputLayer.cuh"

InputLayer::InputLayer(int w, double range, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	scalar = 1 / range;
	for (int i = 0; i < currentLayerNeurons; i++) {
		PassiveNeuron n = PassiveNeuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Passive Neuron " << i << endl;
	}
}

InputLayer::~InputLayer() {
	// TODO Auto-generated destructor stub
}

vector<double> InputLayer::feedforward(vector<double> input) {
	vector<double> output(neurons.size());

	double *deviceInput, *deviceOutput, *deviceScalar;
	PassiveNeuron *deviceNeurons;

	// copy memory to device
	if (hipMalloc((void **)&deviceInput, (input.size() * sizeof(double))) != 0) cout << "error 1" << endl;
	if (hipMalloc((void **)&deviceOutput, (neurons.size() * sizeof(double))) != 0) cout << "error 2" << endl;
	if (hipMalloc((void **)&deviceScalar, sizeof(double)) != 0) cout << "error 3" << endl;
	if (hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(PassiveNeuron))) != 0) cout << "error 4" << endl;

	if (hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice) != 0) cout << "error 5" << endl;
	if (hipMemcpy(&deviceScalar[0], &scalar, sizeof(double), hipMemcpyHostToDevice) != 0) cout << "error 6" << endl;
	if (hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(PassiveNeuron)), hipMemcpyHostToDevice) != 0) cout << "error 7" << endl;

	// start cuda kernel
	hipDeviceSynchronize();
	activateInputNeuron<<<dim3(1, 1), dim3(neurons.size(), 1)>>>(deviceInput, deviceNeurons, deviceScalar, deviceOutput);
	hipDeviceSynchronize();

	// get the output from the device
	if (hipMemcpy(&output[0], &deviceOutput[0], (neurons.size() * sizeof(double)), hipMemcpyDeviceToHost) != 0) cout << "error__" << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if (hipFree(deviceInput) != 0) cout << "error 8" << endl;
	if (hipFree(deviceOutput) != 0) cout << "error 9" << endl;
	if (hipFree(deviceScalar) != 0) cout << "error 10" << endl;
	if (hipFree(deviceNeurons) != 0) cout << "error** 11" << endl;

	return output;
}

vector<double> InputLayer::backpropagate(vector<double> error, double learningRate) {
	return error;
}

