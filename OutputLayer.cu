/*
 * OutputLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "OutputLayer.cuh"

OutputLayer::OutputLayer(int w, int d, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	previousLayerNeurons = d;

	// add neurons to this layer
	for (int i = 0; i < w; i++) {
		Neuron n = Neuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Neuron created " << i << endl;
	} for (int i = 0; i < (w * d); i++) {
		Synapse s = Synapse();
		s.index = i;
		synapses.push_back(s);
		if (debug) cout << "Synapse created " << i << endl;
	}
}

OutputLayer::~OutputLayer() {
	// TODO Auto-generated destructor stub
}

vector<double> OutputLayer::feedforward(vector<double> input) {
	vector<double> sum(currentLayerNeurons), output(currentLayerNeurons);	// variables to store data for math operations

	double *deviceInput, *deviceOutput, *deviceSum, *deviceActivation;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons;

	// copy memory to device
	if (hipMalloc((void **)&deviceInput, (input.size() * sizeof(double))) != 0) cout << "error 1" << endl;
	if (hipMalloc((void **)&deviceOutput, (synapses.size() * sizeof(double))) != 0) cout << "error 2" << endl;
	if (hipMalloc((void **)&deviceSum, (neurons.size() * sizeof(double))) != 0) cout << "error 3" << endl;
	if (hipMalloc((void **)&deviceActivation, (neurons.size() * sizeof(double))) != 0) cout << "error 4" << endl;
	if (hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse))) != 0) cout << "error 5" << endl;
	if (hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron))) != 0) cout << "error 6" << endl;

	if (hipMemcpy(deviceInput, &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice) != 0) cout << "error 7" << endl;
	if (hipMemcpy(deviceSynapses, &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice) != 0) cout << "error 8" << endl;
	if (hipMemcpy(deviceNeurons, &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice) != 0) cout << "error 9" << endl;
	if (hipMemset(deviceSum, 0, (neurons.size() * sizeof(double))) != 0) cout << "error 10" << endl;

	activateSynapse<<<dim3(1, 1), dim3(currentLayerNeurons, previousLayerNeurons)>>>(deviceInput, deviceSynapses, deviceOutput);
	sumInputFromSynapse<<<dim3(1, 1), dim3(currentLayerNeurons, previousLayerNeurons)>>>(deviceOutput, deviceSum);
	activateNeuron<<<dim3(1, 1), dim3(currentLayerNeurons, 1)>>>(deviceSum, deviceNeurons, deviceActivation);

	// get the output from the device
	if (hipMemcpy(&output[0], &deviceActivation[0],(neurons.size() * sizeof(double)), hipMemcpyDeviceToHost) != 0) cout << "error 11" << endl;
	return output;
}

vector<double> OutputLayer::backpropagate(vector<double> error, double learningRate, vector<Neuron> previousLayer) {
	// iterate through each synapse connected to the previous layer
	vector<double> sum(previousLayerNeurons);

	double *deviceError, *deviceSum, deviceLearningRate;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons, *devicePreviousLayer;

	hipMalloc((void **)&deviceError, (error.size() * sizeof(double)));
	hipMalloc((void **)&deviceSum, (previousLayerNeurons * sizeof(double)));
	hipMalloc((void **)&deviceLearningRate, sizeof(double));	// this may be a problem since is only one variable not array
	hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)));
	hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)));
	hipMalloc((void **)&devicePreviousLayer, (previousLayer.size() * sizeof(Neuron)));

	hipMemcpy(deviceError, &error[0], (error.size() * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(&deviceLearningRate, &learningRate, (error.size() * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(deviceSynapses, &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice);
	hipMemcpy(deviceNeurons, &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice);
	hipMemcpy(devicePreviousLayer, &previousLayer[0], (previousLayer.size() * sizeof(Neuron)), hipMemcpyHostToDevice);
	hipMemset(deviceSum, 0, (previousLayerNeurons * sizeof(double)));

	gradientDescent<<<dim3(1, 1), dim3(currentLayerNeurons, previousLayerNeurons)>>>(deviceError, deviceLearningRate, deviceNeurons, devicePreviousLayer, deviceSynapses);
	sumWeightedError<<<dim3(1, 1), dim3(currentLayerNeurons, previousLayerNeurons)>>>(deviceError, deviceNeurons, deviceSynapses, deviceSum);

	hipMemcpy(&synapses[0], &deviceSynapses[0],(synapses.size() * sizeof(Synapse)), hipMemcpyDeviceToHost);
	hipMemcpy(&sum[0], &deviceSum[0],(previousLayerNeurons * sizeof(double)), hipMemcpyDeviceToHost);
	return sum;
}
