/*
 * OutputLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "OutputLayer.cuh"

OutputLayer::OutputLayer(int w, int d, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	previousLayerNeurons = d;

	// add neurons to this layer
	for (int i = 0; i < w; i++) {
		Neuron n = Neuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Neuron created " << i << endl;
	} for (int i = 0; i < (w * d); i++) {
		Synapse s = Synapse();
		s.index = i;
		synapses.push_back(s);
		if (debug) cout << "Synapse created " << i << endl;
	}
}

OutputLayer::~OutputLayer() {
	// TODO Auto-generated destructor stub
}

vector<double> OutputLayer::feedforward(vector<double> input) {
	vector<double> output(currentLayerNeurons);	// variables to store data for math operations

	double *deviceInput, *deviceOutput, *deviceSum, *deviceActivation;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons;

	// copy memory to device
	int status;
	long long startTime = getNanoSec();
	if ((status = hipMalloc((void **)&deviceInput, (input.size() * sizeof(double)))) != 0) cout << "error o-1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceOutput, (synapses.size() * sizeof(double)))) != 0) cout << "error o-2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSum, (neurons.size() * sizeof(double)))) != 0) cout << "error o-3 " << status << endl;
	if ((status = hipMalloc((void **)&deviceActivation, (neurons.size() * sizeof(double)))) != 0) cout << "error o-4 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error o-5 " << status << endl;
	if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)))) != 0) cout << "error o-6 " << status << endl;

	if ((status = hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error o-7 " << status << endl;
	if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error o-8 " << status << endl;
	if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error o-9 " << status << endl;
	if ((status = hipMemset(&deviceSum[0], 0, (neurons.size() * sizeof(double)))) != 0) cout << "error o-10 " << status << endl;
	NeuralNetwork::overhead += (getNanoSec() - startTime);

	startTime = getNanoSec();
	KernelAdapter::startSynapseKernel(deviceInput, deviceSynapses, deviceOutput, currentLayerNeurons, previousLayerNeurons);
	KernelAdapter::startSumInputKernel(deviceOutput, deviceSum, currentLayerNeurons, previousLayerNeurons);
	KernelAdapter::startNeuronKernel(deviceSum, deviceNeurons, deviceActivation, currentLayerNeurons);
	NeuralNetwork::computation += (getNanoSec() - startTime);

	// get the output from the device
	startTime = getNanoSec();
	if ((status = hipMemcpy(&output[0], &deviceActivation[0],(neurons.size() * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error o-11 " << status << endl;
	if ((status = hipMemcpy(&neurons[0], &deviceNeurons[0],(neurons.size() * sizeof(Neuron)), hipMemcpyDeviceToHost)) != 0) cout << "error o-12 " << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceInput)) != 0) cout << "error o-13 " << status << endl;
	if ((status = hipFree(deviceOutput)) != 0) cout << "error o-14 " << status << endl;
	if ((status = hipFree(deviceSum)) != 0) cout << "error o-15 " << status << endl;
	if ((status = hipFree(deviceActivation)) != 0) cout << "error o-16 " << status << endl;
	if ((status = hipFree(deviceSynapses)) != 0) cout << "error o-17 " << status << endl;
	if ((status = hipFree(deviceNeurons)) != 0) cout << "error o-18 " << status << endl;
	hipDeviceSynchronize();
	NeuralNetwork::overhead += (getNanoSec() - startTime);

	return output;
}

vector<double> OutputLayer::backpropagate(vector<double> error, double learningRate, vector<Neuron> previousLayer) {
	// iterate through each synapse connected to the previous layer
	vector<double> sum(previousLayerNeurons);	// must be initialized to be read and write

	double *deviceError, *deviceSum;
	Synapse *deviceSynapses;
	Neuron *devicePreviousLayer;

	int status;
	long long startTime = getNanoSec();
	if ((status = hipMalloc((void **)&deviceError, (error.size() * sizeof(double)))) != 0) cout << "error o-1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSum, (previousLayerNeurons * sizeof(double)))) != 0) cout << "error o-2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error o-4 " << status << endl;
	if ((status = hipMalloc((void **)&devicePreviousLayer, (previousLayer.size() * sizeof(Neuron)))) != 0) cout << "error o-6 " << status << endl;

	if ((status = hipMemcpy(&deviceError[0], &error[0], (error.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error o-7 " << status << endl;
	if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error o-9 " << status << endl;
	if ((status = hipMemcpy(&devicePreviousLayer[0], &previousLayer[0], (previousLayer.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error o-11 " << status << endl;
	if ((status = hipMemset(&deviceSum[0], 0, (previousLayerNeurons * sizeof(double))) != 0)) cout << "error o-12 " << status << endl;
	NeuralNetwork::overhead += (getNanoSec() - startTime);

	startTime = getNanoSec();
	KernelAdapter::startOutputLayerGradientDescentKernel(deviceError, learningRate, devicePreviousLayer, deviceSynapses, currentLayerNeurons, previousLayerNeurons);
	KernelAdapter::startOutputLayerSumErrorKernel(deviceError, deviceSynapses, deviceSum, currentLayerNeurons, previousLayerNeurons);
	NeuralNetwork::computation += (getNanoSec() - startTime);

	// get output from device
	startTime = getNanoSec();
	if ((status = hipMemcpy(&synapses[0], &deviceSynapses[0],(synapses.size() * sizeof(Synapse)), hipMemcpyDeviceToHost)) != 0) cout << "error o-13 " << status << endl;
	if ((status = hipMemcpy(&sum[0], &deviceSum[0],(previousLayerNeurons * sizeof(double)), hipMemcpyDeviceToHost)) != 0)cout << "error o-14 " << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceError)) != 0) cout << "error test o-15 " << status << endl;
	if ((status = hipFree(deviceSum)) != 0)cout << "error o-16 " << status << endl;
	if ((status = hipFree(deviceSynapses)) != 0) cout << "error o-17 " << status << endl;
	if ((status = hipFree(devicePreviousLayer)) != 0) cout << "error o-19 " << status << endl;
	hipDeviceSynchronize();
	NeuralNetwork::overhead += (getNanoSec() - startTime);

	return sum;
}
