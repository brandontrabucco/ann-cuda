/*
 * OutputLayer.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "OutputLayer.cuh"

OutputLayer::OutputLayer(int w, int d, bool db) {
	// TODO Auto-generated constructor stub
	debug = db;
	currentLayerNeurons = w;
	previousLayerNeurons = d;

	vector<int> temp = factor(currentLayerNeurons);
	kernelGridHeight = temp[0];
	kernelGridWidth = temp[1];

	temp = factor(previousLayerNeurons);
	kernelBlockHeight = temp[0];
	kernelBlockWidth = temp[1];

	// add neurons to this layer
	for (int i = 0; i < w; i++) {
		Neuron n = Neuron();
		n.index = i;
		neurons.push_back(n);
		if (debug) cout << "Neuron created " << i << endl;
	} for (int i = 0; i < (w * d); i++) {
		Synapse s = Synapse();
		s.index = i;
		synapses.push_back(s);
		if (debug) cout << "Synapse created " << i << endl;
	}
}

OutputLayer::~OutputLayer() {
	// TODO Auto-generated destructor stub
}

vector<double> OutputLayer::feedforward(vector<double> input) {
	vector<double> output(currentLayerNeurons);	// variables to store data for math operations

	double *deviceInput, *deviceOutput, *deviceSum, *deviceActivation;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons;

	// copy memory to device
	int status;
	if ((status = hipMalloc((void **)&deviceInput, (input.size() * sizeof(double)))) != 0) cout << "error 1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceOutput, (synapses.size() * sizeof(double)))) != 0) cout << "error 2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSum, (neurons.size() * sizeof(double)))) != 0) cout << "error 3 " << status << endl;
	if ((status = hipMalloc((void **)&deviceActivation, (neurons.size() * sizeof(double)))) != 0) cout << "error 4 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error 5 " << status << endl;
	if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)))) != 0) cout << "error 6 " << status << endl;

	if ((status = hipMemcpy(&deviceInput[0], &input[0], (input.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error 7 " << status << endl;
	if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error 8 " << status << endl;
	if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error 9 " << status << endl;
	if ((status = hipMemset(&deviceSum[0], 0, (neurons.size() * sizeof(double)))) != 0) cout << "error 10 " << status << endl;

	hipDeviceSynchronize();
	activateSynapse<<<dim3(kernelGridWidth, kernelGridHeight), dim3(kernelBlockWidth, kernelBlockHeight)>>>(deviceInput, deviceSynapses, deviceOutput);	// a block represents current layer, thread is previous layer
	hipDeviceSynchronize();
	sumInputFromSynapse<<<dim3(1, 1), dim3(kernelGridWidth, kernelGridHeight)>>>(deviceOutput, deviceSum, previousLayerNeurons);
	hipDeviceSynchronize();
	activateNeuron<<<dim3(1, 1), dim3(kernelGridWidth, kernelGridHeight)>>>(deviceSum, deviceNeurons, deviceActivation);
	hipDeviceSynchronize();

	// get the output from the device
	if ((status = hipMemcpy(&output[0], &deviceActivation[0],(neurons.size() * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error 11 " << status << endl;
	if ((status = hipMemcpy(&neurons[0], &deviceNeurons[0],(neurons.size() * sizeof(Neuron)), hipMemcpyDeviceToHost)) != 0) cout << "error 12 " << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceInput)) != 0) cout << "error 13 " << status << endl;
	if ((status = hipFree(deviceOutput)) != 0) cout << "error 14 " << status << endl;
	if ((status = hipFree(deviceSum)) != 0) cout << "error 15 " << status << endl;
	if ((status = hipFree(deviceActivation)) != 0) cout << "error 16 " << status << endl;
	if ((status = hipFree(deviceSynapses)) != 0) cout << "error 17 " << status << endl;
	if ((status = hipFree(deviceNeurons)) != 0) cout << "error 18 " << status << endl;
	hipDeviceSynchronize();

	return output;
}

vector<double> OutputLayer::backpropagate(vector<double> error, double learningRate, vector<Neuron> previousLayer) {
	// iterate through each synapse connected to the previous layer
	vector<double> sum(previousLayerNeurons);	// must be initialized to be read and write

	double *deviceError, *deviceSum;
	Synapse *deviceSynapses;
	Neuron *deviceNeurons, *devicePreviousLayer;

	int status;
	if ((status = hipMalloc((void **)&deviceError, (error.size() * sizeof(double)))) != 0) cout << "error 1 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSum, (previousLayerNeurons * sizeof(double)))) != 0) cout << "error 2 " << status << endl;
	if ((status = hipMalloc((void **)&deviceSynapses, (synapses.size() * sizeof(Synapse)))) != 0) cout << "error 4 " << status << endl;
	if ((status = hipMalloc((void **)&deviceNeurons, (neurons.size() * sizeof(Neuron)))) != 0) cout << "error 5 " << status << endl;
	if ((status = hipMalloc((void **)&devicePreviousLayer, (previousLayer.size() * sizeof(Neuron)))) != 0) cout << "error 6 " << status << endl;

	if ((status = hipMemcpy(&deviceError[0], &error[0], (error.size() * sizeof(double)), hipMemcpyHostToDevice)) != 0) cout << "error 7 " << status << endl;
	if ((status = hipMemcpy(&deviceSynapses[0], &synapses[0], (synapses.size() * sizeof(Synapse)), hipMemcpyHostToDevice)) != 0) cout << "error 9 " << status << endl;
	if ((status = hipMemcpy(&deviceNeurons[0], &neurons[0], (neurons.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error 10 " << status << endl;
	if ((status = hipMemcpy(&devicePreviousLayer[0], &previousLayer[0], (previousLayer.size() * sizeof(Neuron)), hipMemcpyHostToDevice)) != 0) cout << "error 11 " << status << endl;
	if ((status = hipMemset(&deviceSum[0], 0, (previousLayerNeurons * sizeof(double))) != 0)) cout << "error 12 " << status << endl;
	hipDeviceSynchronize();

	gradientDescent<<<dim3(kernelGridWidth, kernelGridHeight), dim3(kernelBlockWidth, kernelBlockHeight)>>>(deviceError, learningRate, deviceNeurons, devicePreviousLayer, deviceSynapses);
	hipDeviceSynchronize();
	// iterate for each neuron sum in previous layer;
	sumWeightedError<<<dim3(1, 1), dim3(kernelBlockWidth, kernelBlockHeight)>>>(deviceError, deviceNeurons, deviceSynapses, deviceSum, (currentLayerNeurons));

	hipDeviceSynchronize();
	if ((status = hipMemcpy(&synapses[0], &deviceSynapses[0],(synapses.size() * sizeof(Synapse)), hipMemcpyDeviceToHost)) != 0) cout << "error 13 " << status << endl;
	if ((status = hipMemcpy(&sum[0], &deviceSum[0],(previousLayerNeurons * sizeof(double)), hipMemcpyDeviceToHost)) != 0) cout << "error 14 " << status << endl;
	hipDeviceSynchronize();

	// release memory from GPU
	if ((status = hipFree(deviceError)) != 0) cout << "error 15 " << status << endl;
	if ((status = hipFree(deviceSum)) != 0) cout << "error 16 " << status << endl;
	if ((status = hipFree(deviceSynapses)) != 0) cout << "error 17 " << status << endl;
	if ((status = hipFree(deviceNeurons)) != 0) cout << "error 18 " << status << endl;
	if ((status = hipFree(devicePreviousLayer)) != 0) cout << "error 19 " << status << endl;
	hipDeviceSynchronize();

	return sum;
}
